// main.cu
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vector_add(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int N = 100000;
    size_t size = N * sizeof(float);

    // 在主机上分配内存
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // 初始化输入向量
    for (int i = 0; i < N; ++i) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }

    // 在设备上分配内存
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // 将输入数据从主机复制到设备
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // 每个线程处理一个元素
    int threads_per_block = 256;
    int blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;

    // 启动 GPU 内核
    vector_add<<<blocks_per_grid, threads_per_block>>>(d_A, d_B, d_C, N);

    // 将结果从设备复制到主机
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // 检查结果
    for (int i = 0; i < N; ++i) {
        std::cout << h_A[i] << " + " << h_B[i] << " = " << h_C[i] << std::endl;
    }

    // 释放内存
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
